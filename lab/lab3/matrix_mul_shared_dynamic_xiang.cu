
#include <hip/hip_runtime.h>
#include <stdio.h>
// these are just for timing measurments
#include <time.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

/* You should not change the value of DSIZE */
const int DSIZE = 18432;
int block_size = 32;
const float A_val = 3.0f;
const float B_val = 2.0f;


__global__ void mmul( float *A, float *B, float *C, int ds) 
{
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    int idy = threadIdx.y+blockDim.y*blockIdx.y;
    float temp = 0;
    extern __shared__ float shared[];
    for (int i =0; i < ((ds+blockDim.x-1)/blockDim.x); i = i+1) {
            if (idy < ds && (i*blockDim.x+threadIdx.x) < ds)  {shared[threadIdx.y*blockDim.x + threadIdx.x] = A[idy*ds +  i*blockDim.x+threadIdx.x];}
            else  {shared[threadIdx.y*blockDim.x + threadIdx.x] = 0;}
            if (idx < ds && (i*blockDim.x+threadIdx.y) < ds)  {shared[threadIdx.y*blockDim.x+threadIdx.x+blockDim.x*blockDim.x] = B[(i*blockDim.x+threadIdx.y)*ds+idx];}
            else  {shared[threadIdx.y*blockDim.x + threadIdx.x +  blockDim.x * blockDim.x] = 0;}
        __syncthreads();
        for (int k = 0; k < blockDim.x; ++k) {temp += shared[threadIdx.y*blockDim.x+k] * shared[k*blockDim.x+threadIdx.x+blockDim.x*blockDim.x];}
        __syncthreads();
    }
    if (idy < ds && idx < ds) {C[idy*ds + idx] = temp;}
}

int main(int argc, char *argv[])
{

  float *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

  // these are just for timing
  clock_t t0, t1, t2;
  double t1sum=0.0;
  double t2sum=0.0;

  if (argc == 2) {
      block_size = atoi(argv[1]);
      if (block_size <= 0) {
          fprintf(stderr, "Error: block_size should be >= 1\n");
          exit (1);
      }
  }

  // start timing
  t0 = clock();

  h_A = new float[DSIZE*DSIZE];
  h_B = new float[DSIZE*DSIZE];
  h_C = new float[DSIZE*DSIZE];

  for (int i = 0; i < DSIZE*DSIZE; i++){
        h_A[i] = A_val;
        h_B[i] = B_val;
        h_C[i] = 0;
  }
  printf("Init took %f seconds.  Begin compute\n", t1sum);

  // Initialization timing
  t1 = clock();
  t1sum = ((double)(t1-t0))/CLOCKS_PER_SEC;
  printf("Init took %f seconds.  Begin compute\n", t1sum);

  // Allocate device memory and copy input data over to GPU
  hipMalloc(&d_A, DSIZE*DSIZE*sizeof(float));
  cudaCheckErrors("hipMalloc failure");
  hipMalloc(&d_B, DSIZE*DSIZE*sizeof(float));
  cudaCheckErrors("hipMalloc failure");
  hipMalloc(&d_C, DSIZE*DSIZE*sizeof(float));
  cudaCheckErrors("hipMalloc failure");

  hipMemcpy(d_A, h_A, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");
  hipMemcpy(d_B, h_B, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");

  // Cuda processing sequence step 1 is complete

  // Launch kernel
  dim3 block(block_size, block_size);  // dim3 variable holds 3 dimensions
  dim3 grid((DSIZE+block.x-1)/block.x, (DSIZE+block.y-1)/block.y);
  mmul<<<grid, block, sizeof(float)*block_size*block_size*2>>>(d_A, d_B, d_C, DSIZE);
  cudaCheckErrors("kernel launch failure");

  // Cuda processing sequence step 2 is complete
  // Copy results back to host
  hipMemcpy(h_C, d_C, DSIZE*DSIZE*sizeof(float), hipMemcpyDeviceToHost);
  cudaCheckErrors("hipMemcpy D2H failure");

  // GPU timing
  t2 = clock();
  t2sum = ((double)(t2-t1))/CLOCKS_PER_SEC;
  printf ("Done. Compute took %f seconds\n", t2sum);


  for (int i = 0; i < DSIZE*DSIZE; i++) if (h_C[i] != A_val*B_val*DSIZE) {printf("mismatch at index %d, was: %f, should be: %f\n", i, h_C[i], A_val*B_val*DSIZE); return -1;}
  //Free memory
  free(h_A);
  free(h_B);
  free(h_C);
  // free cuda memory;
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  return 0;
}
  
